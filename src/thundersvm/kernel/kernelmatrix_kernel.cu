#include "hip/hip_runtime.h"
//
// Created by jiashuai on 17-9-20.
//
#include <thundersvm/syncarray.h>
#include <hipsparse.h>
#include "thundersvm/kernel/kernelmatrix_kernel.h"
#include <thundersvm/config.h>

#include <hipblas.h>
#include <hipcub/hipcub.hpp>
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;
#define TDEF(x_) std::chrono::high_resolution_clock::time_point x_##_t0, x_##_t1;
#define TSTART(x_) x_##_t0 = Clock::now();
#define TEND(x_) x_##_t1 = Clock::now();
#define TPRINT(x_, str) printf("%-20s \t%.6f\t sec\n", str, std::chrono::duration_cast<std::chrono::microseconds>(x_##_t1 - x_##_t0).count()/1e6);
#define TINT(x_) std::chrono::duration_cast<std::chrono::microseconds>(x_##_t1 - x_##_t0).count()
extern long long time1;
extern long long time3;

using namespace hipcub;
namespace svm_kernel {
    
    __global__ void
    kernel_get_working_set_ins(const kernel_type *val, const int *col_ind, const int *row_ptr, const int *data_row_idx,
                               kernel_type *data_rows,
                               int m, int n) {
        KERNEL_LOOP(i, m) {
            int row = data_row_idx[i];
            for (int j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
                int col = col_ind[j];
                data_rows[col * m + i] = val[j]; // col-major for cuSPARSE
            }
            //replace col-major to row-major
            //int row = data_row_idx[i];
            //for (int j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
            //    int col = col_ind[j];
            //    data_rows[i*n+col] = val[j]; // row-major for cuSPARSE
            //}
        }
    }

    __global__ void
    kernel_RBF_kernel(const kernel_type *self_dot0, const kernel_type *self_dot1, kernel_type *dot_product, int m, int n,
                      kernel_type gamma) {
        //m rows of kernel matrix, where m is the working set size; n is the number of training instances
        KERNEL_LOOP(idx, m * n) {
            int i = idx / n;//i is row id
            int j = idx % n;//j is column id
            dot_product[idx] = expf(-(self_dot0[i] + self_dot1[j] - dot_product[idx] * 2) * gamma);
        }
    }

    __global__ void
    kernel_RBF_kernel(const int *self_dot0_idx, const kernel_type *self_dot1, kernel_type *dot_product, int m, int n,
                      kernel_type gamma) {
        //compute m rows of kernel matrix, where m is the working set size and n is the number of training instances, according to idx
        KERNEL_LOOP(idx, m * n) {
            int i = idx / n;//i is row id
            int j = idx % n;//j is column id
            dot_product[idx] = expf(-(self_dot1[self_dot0_idx[i]] + self_dot1[j] - dot_product[idx] * 2) * gamma);
        }
    }

    __global__ void
    kernel_sum_kernel_values(const float_type *coef, int total_sv, const int *sv_start, const int *sv_count,
                             const float_type *rho,
                             const kernel_type *k_mat, float_type *dec_values, int n_classes, int n_instances) {
        KERNEL_LOOP(idx, n_instances) {
            int k = 0;
            int n_binary_models = n_classes * (n_classes - 1) / 2;
            for (int i = 0; i < n_classes; ++i) {
                for (int j = i + 1; j < n_classes; ++j) {
                    int si = sv_start[i];
                    int sj = sv_start[j];
                    int ci = sv_count[i];
                    int cj = sv_count[j];
                    const float_type *coef1 = &coef[(j - 1) * total_sv];
                    const float_type *coef2 = &coef[i * total_sv];
                    const kernel_type *k_values = &k_mat[idx * total_sv];
                    double sum = 0;
                    for (int l = 0; l < ci; ++l) {
                        sum += coef1[si + l] * k_values[si + l];
                    }
                    for (int l = 0; l < cj; ++l) {
                        sum += coef2[sj + l] * k_values[sj + l];
                    }
                    dec_values[idx * n_binary_models + k] = sum - rho[k];
                    k++;
                }
            }
        }
    }

    __global__ void
    kernel_poly_kernel(kernel_type *dot_product, kernel_type gamma, kernel_type coef0, int degree, int mn) {
        KERNEL_LOOP(idx, mn) {
            dot_product[idx] = powf(gamma * dot_product[idx] + coef0, degree);
        }
    }

    __global__ void kernel_sigmoid_kernel(kernel_type *dot_product, kernel_type gamma, kernel_type coef0, int mn) {
        KERNEL_LOOP(idx, mn) {
            dot_product[idx] = tanhf(gamma * dot_product[idx] + coef0);
        }
    }

    void sum_kernel_values(const SyncArray<float_type> &coef, int total_sv, const SyncArray<int> &sv_start,
                           const SyncArray<int> &sv_count, const SyncArray<float_type> &rho,
                           const SyncArray<kernel_type> &k_mat,
                           SyncArray<float_type> &dec_values, int n_classes, int n_instances) {
        SAFE_KERNEL_LAUNCH(kernel_sum_kernel_values, coef.device_data(), total_sv, sv_start.device_data(),
                           sv_count.device_data(), rho.device_data(), k_mat.device_data(), dec_values.device_data(),
                           n_classes, n_instances);

    }

    void
    get_working_set_ins(const SyncArray<kernel_type> &val, const SyncArray<int> &col_ind, const SyncArray<int> &row_ptr,
                        const SyncArray<int> &data_row_idx, SyncArray<kernel_type> &data_rows, int m, int n) {
        SAFE_KERNEL_LAUNCH(kernel_get_working_set_ins, val.device_data(), col_ind.device_data(), row_ptr.device_data(),
                           data_row_idx.device_data(), data_rows.device_data(), m, n);

    }

    void
    RBF_kernel(const SyncArray<kernel_type> &self_dot0, const SyncArray<kernel_type> &self_dot1,
               SyncArray<kernel_type> &dot_product, int m,
               int n,
               kernel_type gamma) {
        SAFE_KERNEL_LAUNCH(kernel_RBF_kernel, self_dot0.device_data(), self_dot1.device_data(),
                           dot_product.device_data(), m, n, gamma);
    }

    void
    RBF_kernel(const SyncArray<int> &self_dot0_idx, const SyncArray<kernel_type> &self_dot1,
               SyncArray<kernel_type> &dot_product, int m,
               int n, kernel_type gamma) {
        SAFE_KERNEL_LAUNCH(kernel_RBF_kernel, self_dot0_idx.device_data(), self_dot1.device_data(),
                           dot_product.device_data(), m, n, gamma);
    }

    void poly_kernel(SyncArray<kernel_type> &dot_product, kernel_type gamma, kernel_type coef0, int degree, int mn) {
        SAFE_KERNEL_LAUNCH(kernel_poly_kernel, dot_product.device_data(), gamma, coef0, degree, mn);
    }

    void sigmoid_kernel(SyncArray<kernel_type> &dot_product, kernel_type gamma, kernel_type coef0, int mn) {
        SAFE_KERNEL_LAUNCH(kernel_sigmoid_kernel, dot_product.device_data(), gamma, coef0, mn);
    }

    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    bool cusparse_init;
    //m for instance; n for get_rows num; k for feature num; nnz for number of nonzero
    void dns_csr_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &csr_val,
                     const SyncArray<int> &csr_row_ptr, const SyncArray<int> &csr_col_ind, int nnz,
                     SyncArray<kernel_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }
        kernel_type one(1);
        kernel_type zero(0);

#if (CUDART_VERSION >= 11000)

        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;
        
#ifdef USE_DOUBLE
        hipDataType data_type = HIP_R_64F;
#else//kernel type is float
        hipDataType data_type = HIP_R_32F;
#endif  

        //set L2 cache Persistence

        // hipStream_t stream;
        // hipStreamCreate(&stream);                                                               

        // //声明设备属性变量
        // hipDeviceProp_t prop; 
        // //获取设备属性                                                                  
        // hipGetDeviceProperties(&prop, 0); 
        // //预留L2持久访问的缓存大小
        // size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
        // //设置预留L2持久访问缓存大小
        // hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);                                  

        // //定义持久访问的数据大小
        // size_t window_size = prop.accessPolicyMaxWindowSize;                        
        // //声明CUDA流属性数据结构
        // hipLaunchAttributeValue stream_attribute; 
        // //全局内存数据指针
        // stream_attribute.accessPolicyWindow.base_ptr = (void*)(csr_val.device_data());              
        // //持久访问的数据大小
        // stream_attribute.accessPolicyWindow.num_bytes = window_size;                               
        // //缓存命中率
        // stream_attribute.accessPolicyWindow.hitRatio = window_size/csr_val.size();                                        
        // //缓存命中
        // stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;               
        // //缓存未命中
        // stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;                
        
        // // 将上面的属性设置给CUDA流stream
        // hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);

        // hipsparseSetStream(handle,stream);

        hipsparseCreateCsr(&matA, m, k, nnz, (void*)csr_row_ptr.device_data(), (void*)csr_col_ind.device_data(),
                          (void*)csr_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, data_type);
        hipsparseCreateDnMat(&matB, n, k, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_COL);
        hipsparseCreateDnMat(&matC, m, n, m, (void*)result.device_data(), data_type, HIPSPARSE_ORDER_COL);
        
        // hipEvent_t start_event, stop_event;
        // float cuda_elapsed_ms  = 0;
        // hipEventCreate(&start_event);
        // hipEventCreate(&stop_event);
        // hipEventRecord(start_event, NULL);
        
        size_t buffer_size = 0;
        hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                               &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1,
                               &buffer_size);

        void *p_buffer = nullptr;
        
        hipMalloc((void**)&p_buffer, buffer_size);
        
        hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                   &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                   &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);


        // //将内存访问大小设为0，禁用持久访问
        // stream_attribute.accessPolicyWindow.num_bytes = 0;                                          
        // // 覆盖CUDA流的访问策略属性
        // hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);   
        // //删除L2中的所有持久性行
        // cudaCtxResetPersistingL2Cache();
        // hipStreamDestroy(stream);

        // hipEventRecord(stop_event, NULL);
        // hipEventSynchronize(stop_event);
        // hipEventElapsedTime(&cuda_elapsed_ms, start_event,stop_event);
        // LOG(INFO)<<"csr dns mul time is "<<cuda_elapsed_ms;


        //graph capture
        // hipGraph_t     graph;
        // hipStream_t    stream;
        // hipGraphExec_t graph_exec;
        // hipStreamCreate(&stream);
        // hipsparseSetStream(handle, stream);
        // hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

        // hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        //            &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);
       
        // hipStreamEndCapture(stream, &graph);
        // hipDeviceSynchronize();
        // hipGetLastError();
        // hipGraphInstantiateWithFlags(&graph_exec, graph, 0);

        // //==========================================================================
        // // GRAPH EXECUTION
        // //==========================================================================

        // hipGraphLaunch(graph_exec, stream);

        // // destroy graph
        // hipDeviceSynchronize();
        // hipGraphExecDestroy(graph_exec);
        // hipGraphDestroy(graph);
        // hipStreamDestroy(stream);

        
        hipFree(p_buffer);
        
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);


        //test ror major

        //store tmp result and tmp trans dense_mat
        // hipsparseDnMatDescr_t matB2,matC2;
        // // LOG(INFO)<<"m is "<<m<<" n is "<<n;
        // SyncArray<kernel_type> tmp_result(m*n);
        // SyncArray<kernel_type> tmp_dense1(n*k);
        // SyncArray<kernel_type> tmp_dense2(k*n);

        // tmp_dense2.copy_from(dense_mat);
        
        // kernel_type* h_tmp1 = tmp_dense1.host_data();
        // kernel_type* h_tmp2 = tmp_dense2.host_data();

        // for(int i=0;i<n;i++){
        //     for(int j=0;j<k;j++){
        //         h_tmp1[i*k+j] = h_tmp2[j*n+i];
        //     }

        // }
        // hipsparseCreateDnMat(&matB2, n, k, k, (void*)tmp_dense1.device_data(), data_type, HIPSPARSE_ORDER_ROW);//HIPSPARSE_ORDER_ROW
        // hipsparseCreateDnMat(&matC2, m, n, n, (void*)tmp_result.device_data(), data_type, HIPSPARSE_ORDER_ROW);
        // //test row major
        // size_t buffer_size2 = 0;
        // hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        //                         &one, matA, matB2, &zero, matC2, data_type, HIPSPARSE_SPMM_CSR_ALG2,
        //                         &buffer_size2);

        // void *p_buffer2 = nullptr;
        // g_allocator.DeviceAllocate(&p_buffer2, buffer_size2);
        
        // hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        //             &one, matA, matB2, &zero, matC2, data_type, HIPSPARSE_SPMM_CSR_ALG2, p_buffer2);


        // g_allocator.DeviceFree(p_buffer2);
        // hipsparseDestroySpMat(matA);
        // hipsparseDestroyDnMat(matB2);
        // hipsparseDestroyDnMat(matC2);



        
        //check result
        // kernel_type* h_res = result.host_data();
        // kernel_type* h_tmp = tmp_result.host_data();
        // kernel_type diff = 0;
        // for(int i = 0;i<n;i++){
        //     for(int j= 0;j<m;j++){
        //         h_res[i*m+j] = h_tmp[j*n+i];
        //         //diff+=fabs(h_res[i*m+j]-h_tmp[j*n+i]);
        //         //LOG(INFO)<<"result is "<<h_res[i*m+j]<<" tmp res is "<<h_tmp[j*n+i];
        //     }
        // }
        // //LOG(INFO)<<"diff is "<<diff;

        // result.device_data();
            
        
        //try cub SpMV
        
        //def
       //SyncArray<kernel_type> d_values_;
       //SyncArray<int> d_column_indices_;//index of each value of all the instances
       //SyncArray<int> d_row_offsets_;//the start positions of the instances
       //SyncArray<kernel_type> d_mat_x_;
       //SyncArray<kernel_type> d_mat_xt;
       ////kernel_type *d_vector_y = result.device_data();

       ////resize
       //d_values_.resize(csr_val.size());
       //d_column_indices_.resize(csr_col_ind.size());
       //d_row_offsets_.resize(csr_row_ptr.size());
       //d_mat_x_.resize(dense_mat.size());
       //d_mat_xt.resize(dense_mat.size());
       ////copy
       //d_values_.copy_from(csr_val );
       //d_column_indices_.copy_from(csr_col_ind);
       //d_row_offsets_.copy_from(csr_row_ptr);
       //d_mat_x_.copy_from(dense_mat);
       //d_mat_xt.copy_from(dense_mat);


       //kernel_type *d_values = d_values_.device_data();
       //int *d_row_offsets = d_row_offsets_.device_data();
       //int *d_column_indices = d_column_indices_.device_data();

       //kernel_type* h_mat_x = d_mat_x_.host_data();
       //kernel_type* h_mat_xt = d_mat_xt.host_data();

       ////trans

       //TDEF(trans)
       //TSTART(trans)
       //for(int i=0;i<n;i++){
       //    for(int j=0;j<k;j++){
       //        h_mat_x[i*k+j] = h_mat_xt[j*n+i];
       //    }

       //}
       //TEND(trans)
       //time1+=TINT(trans);
       ////for(int i = 0;i<5;i++){
       ////    LOG(INFO)<<dense_mat.host_data()[i*n+1]<<" "<<h_mat_x[i+1*k];
       ////}
       ////LOG(INFO)<<"done";


       //kernel_type *d_mat_x = d_mat_x_.device_data();
       //
       //// kernel_type *d_vector_y;
       ////hipMalloc((void**)&d_vector_y,m*n*sizeof(kernel_type));
       //// hipMallocManaged((void**)&d_vector_y,m*n*sizeof(kernel_type));
       //    
       //SyncArray<kernel_type> tmp_res(m*n);
       //kernel_type *d_vector_y = tmp_res.device_data();
       ////kernel_type *h_tmp = tmp_res.host_data();
       //void* d_temp_storage = NULL;
       //size_t temp_storage_bytes = 0;
       //hipcub::DeviceSpmv::CsrMV(d_temp_storage, temp_storage_bytes, d_values,
       //                        d_row_offsets, d_column_indices, d_mat_x, d_vector_y,
       //                        m, k, nnz);

       //g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
       //
       //// hipcub::DeviceSpmv::CsrMV(d_temp_storage, temp_storage_bytes, d_values,
       ////                         d_row_offsets, d_column_indices, d_mat_x, d_vector_y,
       ////                         m, k, nnz);
       //


       // for(int i=0;i<n;i++){

       //    hipcub::DeviceSpmv::CsrMV( d_temp_storage, temp_storage_bytes, d_values,
       //                            d_row_offsets, d_column_indices, d_mat_x+i*k, d_vector_y+i*m,
       //                            m, k, nnz);
       //    
       // }
       // 
       // hipDeviceSynchronize();
       // //copy 
       // result.copy_from(tmp_res);
       // //check
       // // kernel_type *h_res = result.host_data();
       // // kernel_type *h_tmp = tmp_res.host_data();
       // // //
       // // float r = 0;
       // // for(int i = 0;i<n*k;i++){
       // //    // r+=fabs(h_tmp[i]);
       // //    r +=fabs(h_res[i]-h_tmp[i]);
       // // }
       // // LOG(INFO)<<"n is "<<n<<" k is "<<k<<" res sum is "<<r;
       // 
       // g_allocator.DeviceFree(d_temp_storage);
        


        
        
#else

#ifdef USE_DOUBLE
        hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                        m, n, k, nnz, &one, descr, csr_val.device_data(), csr_row_ptr.device_data(),
                        csr_col_ind.device_data(),
                        dense_mat.device_data(), n, &zero, result.device_data(), m);
#else//kernel type is float
        hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                        m, n, k, nnz, &one, descr, csr_val.device_data(), csr_row_ptr.device_data(),
                        csr_col_ind.device_data(),
                        dense_mat.device_data(), n, &zero, result.device_data(), m);

        //hipsparseScsrmm return row-major matrix, so no transpose is needed
#endif // ifdef USE_DOUBLE

#endif // if CUDART_VERSION >= 11000
    }



    //dns dns mul

    hipblasHandle_t handle_blas;
    bool cublas_init;
    void dns_dns_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_a,const SyncArray<kernel_type> &dense_b,kernel_type beta, 
                     SyncArray<kernel_type> &result){

        if (!cublas_init) {
            hipblasCreate(&handle_blas);
            cublas_init = true;
        }

        kernel_type alpha=1.0;
        const kernel_type* d_dense_a = dense_a.device_data();
        const kernel_type* d_dense_b = dense_b.device_data();

        hipblasSgemm(handle_blas,HIPBLAS_OP_T,HIPBLAS_OP_N, m, n, k,&alpha,dense_a.device_data(), k, dense_b.device_data(), k,&beta, result.device_data(), m);
        

    }

    //csr csr mul
    void csr_csr_mul_cuda(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &csr_val,
                     const SyncArray<int> &csr_row_ptr, const SyncArray<int> &csr_col_ind, int nnz,
                     SyncArray<kernel_type> &result){

        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }

        
        kernel_type alpha(1);
        kernel_type beta(0);
        hipDataType data_type = HIP_R_32F;
        hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        hipsparseSpMatDescr_t matA, matB, matC;
        
        hipsparseDnMatDescr_t tmp_mat,result_mat;

        

        void* dBuffer = NULL;
        size_t bufferSize = 0;

        void*  dBuffer1 = NULL, *dBuffer2 = NULL,*dBuffer3 = NULL;
        size_t bufferSize1 = 0, bufferSize2 = 0 ,bufferSize3= 0;

        

        int *tmp_csr_row;
        hipMalloc((void**) &tmp_csr_row,(k + 1) * sizeof(int));

        //create matrix
        hipsparseCreateCsr(&matA, m, k, nnz, (void*)csr_row_ptr.device_data(), (void*)csr_col_ind.device_data(),
                          (void*)csr_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateCsr(&matB, k, n, 0,
                                      tmp_csr_row, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateCsr(&matC, m, n, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateDnMat(&result_mat, m, n, m, (void*)result.device_data(),
                                        data_type, HIPSPARSE_ORDER_COL);

        
     
        //dense转化为csr格式, shape k*n
        
        hipsparseCreateDnMat(&tmp_mat, k, n, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_ROW);

        hipsparseDenseToSparse_bufferSize(
                                        handle, tmp_mat, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize);
        hipMalloc((void**)&dBuffer, bufferSize);


        hipsparseDenseToSparse_analysis(handle, tmp_mat, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer);

        int64_t num_rows_tmp, num_cols_tmp, nnz_tmp;
        int *d_csr_columns;
        
        kernel_type* d_csr_values;

        hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,&nnz_tmp);
        
        hipMalloc((void**) &d_csr_columns, nnz_tmp * sizeof(int));
        hipMalloc((void**) &d_csr_values,  nnz_tmp * sizeof(kernel_type));
        

        hipsparseCsrSetPointers(matB, tmp_csr_row, d_csr_columns,d_csr_values);
        hipsparseDenseToSparse_convert(handle, tmp_mat, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer);


        //timing
        hipEvent_t start_event, stop_event;
        float cuda_elapsed_ms  = 0;
        hipEventCreate(&start_event);
        hipEventCreate(&stop_event);
        hipEventRecord(start_event, NULL);
        //csr csr mul

        hipsparseSpGEMMDescr_t spgemmDesc;
        hipsparseSpGEMM_createDescr(&spgemmDesc);

        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      data_type, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL);
        hipMalloc((void**) &dBuffer1, bufferSize1);

        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      data_type, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1);

        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               data_type, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL);

        hipMalloc((void**) &dBuffer2, bufferSize2);

        hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           data_type, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2);

        int64_t C_num_rows1, C_num_cols1, C_nnz1;
        hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,&C_nnz1);

        int *dC_csrOffsets,*dC_columns;
        kernel_type* dC_values;

        hipMalloc((void**) &dC_csrOffsets, (m+1) * sizeof(int));
        hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int));
        hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(kernel_type));
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values);
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);

        hipEventRecord(stop_event, NULL);
        hipEventSynchronize(stop_event);
        hipEventElapsedTime(&cuda_elapsed_ms, start_event,stop_event);
        LOG(INFO)<<"csr csr mul time is "<<cuda_elapsed_ms;
        //csr to dns

        hipsparseSparseToDense_bufferSize(
                                        handle, matC, result_mat,
                                        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                        &bufferSize3);
        hipMalloc(&dBuffer3, bufferSize3);
        
        hipsparseSparseToDense(handle, matC, result_mat,
                                          HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                          dBuffer3);
        
        

        hipsparseDestroySpMat(matA);
        hipsparseDestroySpMat(matB);
        hipsparseDestroySpMat(matC);
        hipsparseSpGEMM_destroyDescr(spgemmDesc);
        hipsparseDestroyDnMat(result_mat);
        hipFree(dBuffer);
        hipFree(dBuffer1);
        hipFree(dBuffer2);
        hipFree(dBuffer3);
        hipFree(tmp_csr_row);
        hipFree(d_csr_columns);
        hipFree(d_csr_values);
        hipsparseDestroyDnMat(tmp_mat);
        hipFree(dBuffer);
        hipFree(dC_csrOffsets);
        hipFree(dC_columns);
        hipFree(dC_values);




    } 


    //bsr dns mul

    void bsr_dns_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &bsr_val,
                     const SyncArray<int> &bsr_row_ptr, const SyncArray<int> &bsr_col_ind, 
                     SyncArray<kernel_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }
        kernel_type alpha(1);
        kernel_type beta(0);

        hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

        int nnzb = bsr_col_ind.size();
        int mb = bsr_row_ptr.size()-1;
        int blockSize = sqrt(bsr_val.size()/nnzb);
        int nb = (k+blockSize-1)/blockSize;

        //mul
        hipsparseSbsrmm(handle,
               dir,
               HIPSPARSE_OPERATION_NON_TRANSPOSE,
               HIPSPARSE_OPERATION_TRANSPOSE,
               mb, n, nb, nnzb, &alpha,
               descr, bsr_val.device_data(), bsr_row_ptr.device_data(), bsr_col_ind.device_data(), blockSize,
               dense_mat.device_data(), n,
               &beta, result.device_data(), m);
    }



    void csc_dns_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &csc_val,
                     const SyncArray<int> &csc_row_ptr, const SyncArray<int> &csc_col_ind, int nnz,
                     SyncArray<kernel_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }
        
        kernel_type one(1);
        kernel_type zero(0);

        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;

        hipDataType data_type = HIP_R_32F;

        // hipsparseCreateCsr(&matA, k, m, nnz, (void*)csc_col_ind.device_data(), (void*)csc_row_ptr.device_data(),
        //                   (void*)csc_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        //                   HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateCsc(&matA, m, k, nnz, (void*)csc_col_ind.device_data(), (void*)csc_row_ptr.device_data(),
                          (void*)csc_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, data_type);
        hipsparseCreateDnMat(&matB, n, k, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_COL);
        hipsparseCreateDnMat(&matC, m, n, m, (void*)result.device_data(), data_type, HIPSPARSE_ORDER_COL);

        size_t buffer_size = 0;
        hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                                &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1,
                                &buffer_size);

        void *p_buffer = nullptr;
        hipMalloc((void**)&p_buffer, buffer_size);

        hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                   &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                    &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);

        hipFree(p_buffer);
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);
    }

}


